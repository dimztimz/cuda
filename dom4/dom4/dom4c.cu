#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

void checkCudaError(hipError_t err);

__global__ void cestotaBezAtomic(int * buckets, int bucketsLen, int totalThreads)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= totalThreads) {
		return;
	}
	int bid = tid % bucketsLen;
	buckets[bid] += 1;

}

__global__ void cestota(int * buckets, int bucketsLen, int totalThreads)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= totalThreads) {	
		return;
	}
	int bid = tid % bucketsLen;
	atomicAdd(&buckets[bid], 1);

}

int main(int argc, char* argv[])
{
	int bucketsSize, totalThreads, threadsPerBlock;
	int * hostBuckets, * devBuckets;

	if (argc < 4) {
		bucketsSize = 10;
		totalThreads = 10000;
		threadsPerBlock = 100;
	} else {
		sscanf(argv[1], "%d", &totalThreads);
		sscanf(argv[2], "%d", &threadsPerBlock);
		sscanf(argv[3], "%d", &bucketsSize);
	}

	hostBuckets = (int*)calloc(bucketsSize, sizeof(int));
	checkCudaError(hipMalloc(&devBuckets, sizeof(int) * bucketsSize));

	hipEvent_t start, end;
	float vreme;
	hipEventCreate(&start);
	hipEventCreate(&end);

	int numBlocks = totalThreads / threadsPerBlock + (totalThreads%threadsPerBlock != 0);

	checkCudaError(hipMemset(devBuckets, 0, sizeof(int) * bucketsSize));
	hipEventRecord(start);
	cestota<<<numBlocks, threadsPerBlock>>>(devBuckets, bucketsSize, totalThreads);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&vreme, start, end);
	hipMemcpy(hostBuckets, devBuckets, sizeof(int)*bucketsSize, hipMemcpyDeviceToHost);
	printf("So atomic:\telementi=%d blokovi=%d niski=%d koficki=%d vreme=%f\n", totalThreads, numBlocks, threadsPerBlock, bucketsSize, vreme);

	checkCudaError(hipMemset(devBuckets, 0, sizeof(int) * bucketsSize));
	hipEventRecord(start);
	cestotaBezAtomic<<<numBlocks, threadsPerBlock>>>(devBuckets, bucketsSize, totalThreads);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&vreme, start, end);
	hipMemcpy(hostBuckets, devBuckets, sizeof(int)*bucketsSize, hipMemcpyDeviceToHost);
	printf("Bez atomic:\telementi=%d blokovi=%d niski=%d koficki=%d vreme=%f\n", totalThreads, numBlocks, threadsPerBlock, bucketsSize, vreme);

	
	
	/*for(int i = 0; i<bucketsSize; i++) {
		printf("%d ", hostBuckets[i]);
	}
	printf("\n");*/

	return 0;
}

void checkCudaError(hipError_t err)
{
	if (err != hipSuccess)
		printf("%s\n", hipGetErrorString(err));
}