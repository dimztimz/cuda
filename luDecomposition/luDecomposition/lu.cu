#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>

#include <time.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>

using namespace std;

template <typename T>
__global__ void luStaro(T * matrica, T * lMat, T * uMat, int n)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (idx >= n*n) {
		return;
	}

	int redica = idx / n;
	int kolona = idx % n;

	uMat[idx] = matrica[idx];

	lMat[idx] = 0;
	for (int i = 0; i < n; i++) {
		// presmetaj koeficienti
		
		/*
		//so sledniov kod prvite n posledovatelni niski od celata mreza zapisuvaat vo edna kolona od L
		if (idx < i) { //vo ovoj if vleguvaat prvite i niski od celata mreza t.e. od 0 do i-1
			lMat[idx * n + i] = 0;
		} else if (idx < n) { // a vo ovoj ostanatite do n t.e od i do n-1
			lMat[idx * n + i] = uMat[idx * n + i] / uMat[i * n + i];
			//printf("%f ", lMat[idx * n + i]);
		}
		*/
		if (kolona == i && redica >= i) {
			lMat[redica*n + kolona] = uMat[redica*n + kolona] / uMat[kolona*n + kolona];
		}
		//problemot e tuka, nekoi blokovi pocnuva eliminacija pred da se presmeta koeficietot koj go stavame vo L
		//na mestovo kade st ostoi komentarov idealno treba da stoi bariera na celata mreza, a ne samo na vo blok
		//edno resenie da presmetuvame koef vo L, a potoa sekoja niska posebno uste pri samata eliminacija da si presmetuva

		__threadfence(); // ne vrsi rabota
		//eliminacija
		if (redica > i) {
			uMat[redica * n + kolona] -= lMat[redica * n + i] * uMat[i * n + kolona];
		}
		
		__threadfence();

	}
}

template <typename T>
__global__ void lu1(T * L, T * U, int n, int i)
{
	//vo ovoj cekor presmetivame koef so koj kje mnozime gorna redica i kje gi eliminrame redicite pod nea
	//idx ni e redica pocnuvanjki od A[i], i ni e kolona
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int redica = idx + i;
	if (redica < n) {
		L[redica*n + i] = U[redica*n + i] / U[i*n + i];
	}
}

template <typename T>
__global__ void lu2(T * L, T * U, int n, int i)
{
	//ovde idx ni e globalen indeks na niskata i globalen indeks vo matricata (row major) pocnuvajki od U[i+1][i]
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx >= (n-i-1)*(n-i)) {
		return;
	}
	//(n - i) sirina na redica sto ja obrabotivame. delime so sorinata i gi dodavame pomestuvanjeata t.e. offsetite
	int redica = idx / (n - i) + i + 1;
	int kolona = idx % (n - i) + i;
	U[redica * n + kolona] -= L[redica * n + i] * U[i * n + kolona];
}

template <typename T>
void printMatrix(T * mat, int red, int kol)
{
	for (int i = 0; i < red; i++) {
		for (int j = 0; j < red; j++) {
			cout << mat[i*kol + j] << ' ';
		}
		cout << endl;
	}
}

template <typename T>
void lu(T * d_L, T * d_U, int n)
{
	const int NUMTHREAD = 128;

	float * L = new float[n*n];
	float * U = new float[n*n];

	/*hipMemcpy(U, d_U, sizeof(U[0])*n*n, hipMemcpyDeviceToHost);
	printMatrix<float>(U, n, n);
	cout << "======" << endl;*/

	hipMemset(d_L, 0, sizeof(T)*n*n);
	for (int i = 0; i < n-1; i++) {
		int lu1blocks = (n-i + NUMTHREAD - 1) / NUMTHREAD;
		int lu2blocks = ((n-i-1)*(n-i) + NUMTHREAD - 1) / NUMTHREAD;
		lu1<T> << <lu1blocks, NUMTHREAD >> >(d_L, d_U, n, i);
		lu2<T> << <lu2blocks, NUMTHREAD >> >(d_L, d_U, n, i);

		/*hipMemcpy(L, d_L, sizeof(L[0])*n*n, hipMemcpyDeviceToHost);
		hipMemcpy(U, d_U, sizeof(U[0])*n*n, hipMemcpyDeviceToHost);

		printMatrix<float>(L, n, n);
		cout << '*' << endl;
		printMatrix<float>(U, n, n);
		cout << "======" << endl;*/
	}
	lu1<T> << <1, 1 >> >(d_L, d_U, n, n-1);

	delete[] L; delete[] U;
}

void matMulCpuNaive(const float * A, const float * B, float * C, int redA, int kolARedB, int kolB)
{
	for (int i = 0; i < redA; i++) {
		for (int j = 0; j< kolB; j++) {
			float s = 0.0f;
			const float * Apok = A + i*kolARedB;
			const float * Bpok = B + j;
			for (int k = 0; k<kolARedB; k++) {
				s += *Apok * *Bpok;
				Apok++;
				Bpok += kolB;
			}
			C[i * kolB + j] = s;
		}
	}
}

bool proveriGreska(const float * referenceMat, const float * presmetanaMat, int len)
{
	bool greska = false;
	for (int i = 0; i<len; i++) {
		if (referenceMat[i] != presmetanaMat[i]) {
			greska = true;
			//printf("Greska vo index %d, %f != %f\n", i, referenceMat[i], presmetanaMat[i]);
		}
	}
	return greska;
}

bool test1()
{
	float A[16] = { 1, 2, 3, 4, 4, 5, 6, 7, 7, 8, 10, 11, 11, 12, 13, 15 };
	float L[16], U[16], novoA[16];
	int n = 4;
	float *d_U, *d_L;
	hipMalloc(&d_L, sizeof(L));
	hipMalloc(&d_U, sizeof(U));
	hipMemcpy(d_U, A, sizeof(A), hipMemcpyHostToDevice);
	lu<float>(d_L, d_U, n);
	hipMemcpy(L, d_L, sizeof(L), hipMemcpyDeviceToHost);
	hipMemcpy(U, d_U, sizeof(U), hipMemcpyDeviceToHost);
	hipFree(d_L);
	hipFree(d_U);
	printMatrix<float>(L, 4, 4);
	cout << '*' << endl;
	printMatrix<float>(U, 4, 4);
	cout << '=' << endl;
	matMulCpuNaive(L, U, novoA, 4, 4, 4);
	printMatrix<float>(novoA, 4, 4);
	bool greska = proveriGreska(A, novoA, 16);
	if (greska) {
		cout << "razlicno so referentna matrica, GRESKA" << endl;
		printMatrix<float>(A, 4, 4);
	}
	else {
		cout << "ednakvo so referentna matrica, TOCNO" << endl;
	}
	return greska;
}

bool randomMatrixTest()
{
	srand(time(NULL));
	int n = 6;
	float * A = new float[n*n];
	float * L = new float[n*n];
	float * U = new float[n*n];
	float * novoA = new float[n*n];

	for (int i = 0; i < n*n; i++) {
		A[i] = (float)(rand() % 1000);
	}

	float *d_U, *d_L;
	hipMalloc(&d_L, sizeof(L[0])*n*n);
	hipMalloc(&d_U, sizeof(U[0])*n*n);
	hipMemcpy(d_U, A, sizeof(A[0])*n*n, hipMemcpyHostToDevice);
	lu<float>(d_L, d_U, n);
	hipMemcpy(L, d_L, sizeof(L[0])*n*n, hipMemcpyDeviceToHost);
	hipMemcpy(U, d_U, sizeof(U[0])*n*n, hipMemcpyDeviceToHost);
	hipFree(d_L);
	hipFree(d_U);

	printMatrix<float>(L, n, n);
	cout << '*' << endl;
	printMatrix<float>(U, n, n);
	cout << '=' << endl;
	matMulCpuNaive(L, U, novoA, n, n, n);
	printMatrix<float>(novoA, n, n);
	bool greska = proveriGreska(A, novoA, n*n);
	if (greska) {
		cout << "razlicno so referentna matrica, GRESKA" << endl;
		printMatrix<float>(A, n, n);
	}
	else {
		cout << "ednakvo so referentna matrica, TOCNO" << endl;
	}
	delete[] A;
	delete[] L;
	delete[] U;
	delete[] novoA;
	return greska;
}

int main()
{
	test1();
	cout << endl;
	randomMatrixTest();
	system("pause");
	return 0;
}
