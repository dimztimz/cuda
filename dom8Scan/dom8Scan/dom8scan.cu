#include <cstdio>
#include <cstdlib>
#include <cfloat>
#include <cstring>

#include <hip/hip_runtime.h>
#include <omp.h>

template <int THREADSBLOCK, typename T>
__global__ void inclusiveScanHillisSteele1InBlock(const T * in, T * out, int len)
{
	__shared__ T cache[THREADSBLOCK*2];

	int shIdx = threadIdx.x;
	cache[shIdx] = 0;

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	T val = 0;
	if (idx < len) {
		val = in[idx];
	}
	shIdx += blockDim.x;
	cache[shIdx] = val;

#pragma unroll
	for (int pomestuvanje = 1; pomestuvanje < THREADSBLOCK; pomestuvanje <<= 1) {
		__syncthreads();
		val += cache[shIdx - pomestuvanje];
		__syncthreads();
		cache[shIdx] = val;
	}
	if (idx < len) {
		out[idx] = val;
	}
}

template <int THREADSBLOCK, typename T>
__global__ void inclusiveScanHillisSteele2OutBlock(const T * out, T * outerBlockScan, int len)
{
	__shared__ T cache[THREADSBLOCK*2];

	int shIdx = threadIdx.x;
	cache[shIdx] = 0;

	int idx = blockDim.x * threadIdx.x + blockDim.x - 1;
	T val = 0;
	if (idx < len) {
		val = out[idx];
	}
	shIdx += blockDim.x;
	cache[shIdx] = val;

	T scanVal = val;
#pragma unroll
	for (int pomestuvanje = 1; pomestuvanje < THREADSBLOCK; pomestuvanje <<= 1) {
		__syncthreads();
		scanVal += cache[shIdx - pomestuvanje];
		__syncthreads();
		cache[shIdx] = scanVal;
	}
	outerBlockScan[threadIdx.x] = scanVal - val;
}

template <int THREADSBLOCK, typename T>
__global__ void inclusiveScanHillisSteele3Merge(T * out, const T * outerBlockScan, int len)
{
	__shared__ T toAdd;
	if (threadIdx.x == 0) {
		toAdd = outerBlockScan[blockIdx.x];
	}
	__syncthreads();
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < len) {
		out[idx] += toAdd;
	}
}

template <int THREADSBLOCK, typename T>
__global__ void exclusiveScanBlelloch1InBlock(const T * in, T * out, int len)
{
	__shared__ T cache[THREADSBLOCK];
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;

	T vrednost = 0;
	if (idx < len) {
		vrednost = in[idx];
	}
	cache[tid] = vrednost;
	__syncthreads();

	//upsweep
#pragma unroll
	for(int pomestuvanje = 1; pomestuvanje < THREADSBLOCK; pomestuvanje <<= 1) {
		int iDesno = pomestuvanje * (tid * 2 + 2) - 1;
		if (iDesno < THREADSBLOCK) {
			cache[iDesno] += cache[iDesno-pomestuvanje];
		}
		__syncthreads();
	}
	
	if (threadIdx.x == 0) {
		cache[THREADSBLOCK-1] = 0;
	}

	//downsweep
#pragma unroll
	for(int pomestuvanje = THREADSBLOCK >> 1; pomestuvanje != 0; pomestuvanje >>= 1) {
		int iDesno = pomestuvanje * (tid * 2 + 2) - 1;
		if (iDesno < THREADSBLOCK) {
			T levo = cache[iDesno - pomestuvanje];
			T desno = cache[iDesno];
			cache[iDesno] = levo + desno;
			cache[iDesno - pomestuvanje] = desno;
		}
		__syncthreads();
	}

	if (idx < len) {
		out[idx] = cache[tid];
	}
}

template <int THREADSBLOCK, typename T>
__global__ void exclusiveScanBlelloch2OutBlock(const T * in, const T * out, T * buff, int len)
{
	__shared__ T cache[THREADSBLOCK];
	int tid = threadIdx.x;
	int idx = blockDim.x * threadIdx.x + blockDim.x - 1;
	

	T vrednost = 0;
	if (idx < len) {
		vrednost = out[idx] + in[idx];
	}
	cache[tid] = vrednost;
	__syncthreads();

	//upsweep
#pragma unroll
	for(int pomestuvanje = 1; pomestuvanje < THREADSBLOCK; pomestuvanje <<= 1) {
		int iDesno = pomestuvanje * (tid * 2 + 2) - 1;
		if (iDesno < THREADSBLOCK) {
			cache[iDesno] += cache[iDesno-pomestuvanje];
		}
		__syncthreads();
	}
	
	if (threadIdx.x == 0) {
		cache[THREADSBLOCK-1] = 0;
	}

	//downsweep
#pragma unroll
	for(int pomestuvanje = THREADSBLOCK >> 1; pomestuvanje != 0; pomestuvanje >>= 1) {
		int iDesno = pomestuvanje * (tid * 2 + 2) - 1;
		if (iDesno < THREADSBLOCK) {
			T levo = cache[iDesno - pomestuvanje];
			T desno = cache[iDesno];
			cache[iDesno] = levo + desno;
			cache[iDesno - pomestuvanje] = desno;
		}
		__syncthreads();
	}

	buff[tid] = cache[tid];
}

void cpuInclusiveScan(float * in, float * out, int len)
{
	float suma = 0;
	for (int i = 0; i < len; i++) {
		suma += in[i];
		out[i] = suma;
	}
}

void cpuExclusiveScan(float * in, float * out, int len)
{
	float suma = 0;
	for (int i = 0; i < len; i++) {
		out[i] = suma;
		suma += in[i];
	}
}

void proveriTocnostInclusive() {
	float * h_in, * h_scan, * h_refScan;
	int maxN = 256*10+10;

	h_in = new float[maxN];
	h_scan = new float[maxN];
	h_refScan = new float[maxN];

	srand(2008);
	for(int i = 0; i<maxN; i++) {
		h_in[i] = (rand() % 4000) + 0.5f;
	}
	cpuInclusiveScan(h_in, h_refScan, maxN);

	float * d_in, * d_scan, *d_buffOutBLockScan;
	hipMalloc(&d_in, maxN * sizeof(float));
	hipMalloc(&d_scan, maxN * sizeof(float));
	hipMalloc(&d_buffOutBLockScan, 256 * sizeof(float));
	hipMemcpy(d_in, h_in, maxN * sizeof(float), hipMemcpyHostToDevice);

	inclusiveScanHillisSteele1InBlock<256, float><<<(maxN+255)/256, 256>>>(d_in, d_scan, maxN);
	inclusiveScanHillisSteele2OutBlock<256, float><<<1, 256>>>(d_scan, d_buffOutBLockScan, maxN);
	inclusiveScanHillisSteele3Merge<256, float><<<(maxN+255)/256, 256>>>(d_scan, d_buffOutBLockScan, maxN);
	hipMemcpy(h_scan, d_scan, maxN * sizeof(float), hipMemcpyDeviceToHost);

	for(int i = 0; i<maxN; i++) {
		if (h_refScan[i] != h_scan[i]) {
			printf("Greska vo index %d: %f != %f\n", i, h_refScan[i], h_scan[i]);
			
		}
	}

	hipFree(d_in);
	hipFree(d_scan);
	hipFree(d_buffOutBLockScan);

	delete[] h_in;
	delete[] h_scan;
	delete[] h_refScan;
}

void proveriTocnostExclusive() {
	float * h_in, * h_scan, * h_refScan;
	int maxN = 5000;

	h_in = new float[maxN];
	h_scan = new float[maxN];
	h_refScan = new float[maxN];

	srand(2008);
	for(int i = 0; i<maxN; i++) {
		h_in[i] = (rand() % 4000) + 0.5f;
	}
	cpuExclusiveScan(h_in, h_refScan, maxN);

	float * d_in, * d_scan, *d_buffOutBLockScan;
	hipMalloc(&d_in, maxN * sizeof(float));
	hipMalloc(&d_scan, maxN * sizeof(float));
	hipMalloc(&d_buffOutBLockScan, 256 * sizeof(float));
	hipMemcpy(d_in, h_in, maxN * sizeof(float), hipMemcpyHostToDevice);

	exclusiveScanBlelloch1InBlock<256, float><<<(maxN+255)/256, 256>>>(d_in, d_scan, maxN);
	exclusiveScanBlelloch2OutBlock<256, float><<<1, 256>>>(d_in, d_scan, d_buffOutBLockScan, maxN);
	inclusiveScanHillisSteele3Merge<256, float><<<(maxN+255)/256, 256>>>(d_scan, d_buffOutBLockScan, maxN);
	hipMemcpy(h_scan, d_scan, maxN * sizeof(float), hipMemcpyDeviceToHost);

	for(int i = 0; i<maxN; i++) {
		if (h_refScan[i] != h_scan[i]) {
			printf("Greska vo index %d: %f != %f\n", i, h_refScan[i], h_scan[i]);
			
		}
	}

	hipFree(d_in);
	hipFree(d_scan);
	hipFree(d_buffOutBLockScan);

	delete[] h_in;
	delete[] h_scan;
	delete[] h_refScan;
}

template <int THREADSBLOCK>
void benchmarkInclusiveHillisSteele(FILE * outFile)
{
	float * d_array, * d_scan, * d_buff;
	hipEvent_t start, end;
	float vreme;
	hipEventCreate(&start);
	hipEventCreate(&end);

	fprintf(outFile, "GPU hillis steele scan\n");
	fprintf(outFile, "N;tile width;vreme\n");
	for (int N = 32; N <= THREADSBLOCK*THREADSBLOCK; N+=32) {
		hipMalloc(&d_array, N*sizeof(float));
		hipMalloc(&d_scan, N*sizeof(float));
		hipMalloc(&d_buff, THREADSBLOCK*sizeof(float));
		hipEventRecord(start);
		inclusiveScanHillisSteele1InBlock<THREADSBLOCK, float>
			<<<(N+THREADSBLOCK-1)/THREADSBLOCK, THREADSBLOCK>>>(d_array, d_scan, N);
		if (N > THREADSBLOCK) {
			inclusiveScanHillisSteele2OutBlock<THREADSBLOCK, float><<<1, THREADSBLOCK>>>
				(d_scan, d_buff, N);
			inclusiveScanHillisSteele3Merge<THREADSBLOCK, float>
				<<<(N+THREADSBLOCK-1)/THREADSBLOCK, THREADSBLOCK>>>(d_scan, d_buff, N);
		}
		hipEventRecord(end);
		hipEventSynchronize(end);
		hipEventElapsedTime(&vreme, start, end);
		hipFree(d_array);
		hipFree(d_scan);
		hipFree(d_buff);
		fprintf(outFile, "%d;%d;%f\n", N, THREADSBLOCK, vreme);
	}

	hipEventDestroy(start);
	hipEventDestroy(end);
}

template <int THREADSBLOCK>
void benchmarkExclusiveBlelloch(FILE * outFile)
{
	float * d_array, * d_scan, * d_buff;
	hipEvent_t start, end;
	float vreme;
	hipEventCreate(&start);
	hipEventCreate(&end);

	fprintf(outFile, "GPU blelloch scan\n");
	fprintf(outFile, "N;tile width;vreme\n");
	for (int N = 32; N <= THREADSBLOCK*THREADSBLOCK; N+=32) {
		hipMalloc(&d_array, N*sizeof(float));
		hipMalloc(&d_scan, N*sizeof(float));
		hipMalloc(&d_buff, THREADSBLOCK*sizeof(float));
		hipEventRecord(start);
		exclusiveScanBlelloch1InBlock<THREADSBLOCK, float>
			<<<(N+THREADSBLOCK-1)/THREADSBLOCK, THREADSBLOCK>>>(d_array, d_scan, N);
		if (N > THREADSBLOCK) {
			exclusiveScanBlelloch2OutBlock<THREADSBLOCK, float><<<1, THREADSBLOCK>>>
				(d_array, d_scan, d_buff, N);
			inclusiveScanHillisSteele3Merge<THREADSBLOCK, float>
				<<<(N+THREADSBLOCK-1)/THREADSBLOCK, THREADSBLOCK>>>(d_scan, d_buff, N);
		}
		hipEventRecord(end);
		hipEventSynchronize(end);
		hipEventElapsedTime(&vreme, start, end);
		hipFree(d_array);
		hipFree(d_scan);
		hipFree(d_buff);
		fprintf(outFile, "%d;%d;%f\n", N, THREADSBLOCK, vreme);
	}

	hipEventDestroy(start);
	hipEventDestroy(end);
}

void benchmark(const char * filenamePrefix)
{
	float * h_array, * h_scan;

	FILE * cpuInclusiveOut = stdout, * cpuExclusiveOut = stdout;
	FILE * gpuInclusiveOut = stdout, * gpuExclusiveOut = stdout;
	if (filenamePrefix != NULL) {
		char filename[256];
		strncpy(filename, filenamePrefix, 256);
		strncat(filename, "_cpuIncusive.csv", 256);
		cpuInclusiveOut = fopen(filename, "w");
		strncpy(filename, filenamePrefix, 256);
		strncat(filename, "_cpuExclusive.csv", 256);
		cpuExclusiveOut = fopen(filename, "w");
		strncpy(filename, filenamePrefix, 256);
		strncat(filename, "_gpuInclusive.csv", 256);
		gpuInclusiveOut = fopen(filename, "w");
		strncpy(filename, filenamePrefix, 256);
		strncat(filename, "_gpuExclusive.csv", 256);
		gpuExclusiveOut = fopen(filename, "w");
	}

	fprintf(cpuInclusiveOut, "CPU inclusive scan\n");
	fprintf(cpuInclusiveOut, "N;vreme\n");
	for (int N = 32; N < 512*512; N+=32) {
		h_array = (float *)calloc(N, sizeof(float));
		h_scan = (float *)calloc(N, sizeof(float));

		double vreme = omp_get_wtime();
		cpuInclusiveScan(h_array, h_scan, N);
		vreme = omp_get_wtime() - vreme;
		fprintf(cpuInclusiveOut, "%d;%lf\n", N, vreme*1000.0);

		free(h_array);
		free(h_scan);
	}

	fprintf(cpuExclusiveOut ,"CPU exclusive scan\n");
	fprintf(cpuExclusiveOut, "N;vreme\n");
	for (int N = 32; N < 512*512; N+=32) {
		h_array = (float *)calloc(N, sizeof(float));
		h_scan = (float *)calloc(N, sizeof(float));

		double vreme = omp_get_wtime();
		cpuExclusiveScan(h_array, h_scan, N);
		vreme = omp_get_wtime() - vreme;
		fprintf(cpuExclusiveOut, "%d;%lf\n", N, vreme*1000.0);

		free(h_array);
		free(h_scan);
	}
	
	benchmarkInclusiveHillisSteele<32>(gpuInclusiveOut);
	benchmarkInclusiveHillisSteele<64>(gpuInclusiveOut);
	benchmarkInclusiveHillisSteele<128>(gpuInclusiveOut);
	benchmarkInclusiveHillisSteele<256>(gpuInclusiveOut);
	benchmarkInclusiveHillisSteele<512>(gpuInclusiveOut);
	benchmarkInclusiveHillisSteele<1024>(gpuInclusiveOut);

	benchmarkExclusiveBlelloch<32>(gpuExclusiveOut);
	benchmarkExclusiveBlelloch<64>(gpuExclusiveOut);
	benchmarkExclusiveBlelloch<128>(gpuExclusiveOut);
	benchmarkExclusiveBlelloch<256>(gpuExclusiveOut);
	benchmarkExclusiveBlelloch<512>(gpuExclusiveOut);
	benchmarkExclusiveBlelloch<1024>(gpuExclusiveOut);

	fclose(cpuInclusiveOut);
	fclose(cpuExclusiveOut);
	fclose(gpuInclusiveOut);
	fclose(gpuExclusiveOut);
}

int main(int argc, char * argv[])
{
	if (argc > 1) {
		benchmark(argv[1]);
	} else {
		benchmark((const char *)NULL);
	}
}